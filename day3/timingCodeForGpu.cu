#include <iostream>
#include <math.h>
#include <chrono>
#define size 90000
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, int *b, int *c, int n) 
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}

int main()
{
    int *a, *b, *c;
    int n = size;
    int *d_a, *d_b, *d_c;
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    a = new int[n];
    b = new int[n];
    c = new int[n];

    for (int i = 0; i < n; i++) {
        a[i] = rand() % 90000;
        b[i] = rand() % 90000;
    }

    hipMallocManaged(&d_a, n * sizeof(int));
    hipMallocManaged(&d_b, n * sizeof(int));
    hipMallocManaged(&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    auto start = chrono::high_resolution_clock::now();
    add<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();
    auto end = chrono::high_resolution_clock::now();

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%d\t", c[i]);
    }
    printf("\n");

    auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
    cout << "Time taken by GPU: " << duration.count() << " microseconds" << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}