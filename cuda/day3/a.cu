#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

// Set the number of threads per block to 256, which is a multiple of the
// number of CUDA cores per multiprocessor (32) and smaller than the maximum
// number of threads per block (1024).
#define THREADS_PER_BLOCK 256

// Set the number of blocks to 64, which is the maximum number of blocks that
// can be launched per multiprocessor. This value should be chosen based on
// the number of CUDA cores per multiprocessor and the number of multiprocessors
// on the GPU.
#define BLOCKS 64

__global__ void add_arrays(int *c, const int *a, const int *b, int size)
{
    // Calculate the index of the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure that the index is within the bounds of the arrays
    if (i < size)
    {
        // Add the elements at the current index in the two input arrays    
        c[i] = a[i] + b[i];
    }
}

int main()
{
    const int size = 15000;
    int *a = (int*)malloc(size * sizeof(int));
    int *b = (int*)malloc(size * sizeof(int));
    int *c = (int*)malloc(size * sizeof(int));

    for(int i = 0; i < size; i++)
    {
        a[i] = rand() % 14999;
        b[i] = rand() % 14999;
    }

    int *d_a, *d_b, *d_c;

    // Allocate memory on the device for arrays a, b, and c
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    // Copy arrays a and b to the device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU with the correct number of threads and blocks
    add_arrays<<<BLOCKS, THREADS_PER_BLOCK>>>(d_c, d_a, d_b, size);

    // Copy the result back to the host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for(int i = 0; i < size; i++)
    {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free the memory allocated on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free the memory allocated on the host
    free(a);
    free(b);
    free(c);

    return 0;
}