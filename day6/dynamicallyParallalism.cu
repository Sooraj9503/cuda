#include<stdio.h>
#include<hip/hip_runtime.h>

__device__ int square(int a)
{
    printf("Tread (%d, %d) - squaring value\n", blockIdx.x, threadIdx.x);
    return a * a;
}

__global__ void doubleValues(int* data, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size)
    {
        int value = data[i];
        
        //call square function and print thread ID
        int squared_value = square(value);
        printf("Thread (%d, %d) - squared value: %d\n", blockIdx.x, threadIdx.x, squared_value);
        data[i] = squared_value * 2;
    }
}

int main()
{
    //Allocate memory on host
    int size = 10;
    int* host_data = new int[size];
    int* dev_data;
    hipMalloc((void **)&dev_data, size * sizeof(int));

    //Initialize data on host
    for(int i = 0; i < size; i++)
    {
        host_data[i] = i;
    }

    //Copy data from host to device
    hipMemcpy(dev_data, host_data, size * sizeof(int), hipMemcpyHostToDevice);

    //Launch kernel function
    int threads = 256;
    int blocks = (size + threads -1) / threads;
    doubleValues<<<blocks, threads>>>(dev_data, size);

    //Wait for kernel to finish
    hipDeviceSynchronize();

    //Copy data from device to host
    hipMemcpy(host_data, dev_data, size * sizeof(int), hipMemcpyDeviceToHost);

    //Print result
    for(int i = 0; i < size; i++)
    {
        printf("Result: %d\n", host_data[i]);
    }

    //Free memory
    hipFree(dev_data);
    delete[] host_data;
    return 0;
}