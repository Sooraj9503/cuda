#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__ void add_arrays(int *c, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    const int size = 5;
    int a[size] = {1, 2, 3, 4, 5};
    int b[size] = {1, 2, 3, 4, 5};
    int c[size];

    int *d_a, *d_b, *d_c;

    //Allocate memory on the device for arrays a, b, and c
    hipMalloc((void**)&d_a, size * sizeof(int));
    hipMalloc((void**)&d_b, size * sizeof(int));
    hipMalloc((void**)&d_c, size * sizeof(int));

    //Copy arrays a and b to the device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    //Launch the kernel on the GPU
    add_arrays<<<1, size>>>(d_c, d_a, d_b, size);

    //Copy the result back to the host
    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    //Print the result
    for(int i = 0; i < size; i++)
    {
        printf("%d ", c[i]);
    }
    printf("\n");

    //Free the memory allocated on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}