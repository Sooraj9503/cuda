
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello, World from GPU!\n");
}

int main() {
    hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
