#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#define size 1000000;
using namespace std;

__global__ void prime(int *num, int *p)
{
    int a = 1;
    for (int i = 2; i < num - 1; i++)
    {
        if (num % i == 0)
        {
            a = 0;
            break;
        }
    }
    if (a == 1)
        p[size]= num;
}

int main()
{
    int num = 0;
    cout << "Enter until which number you want prime numbers: " << endl;
    cin >> num;
    //host variables
    int num, p[size];
    //device variables
    int *d_num, *d_p;

    //allocate memory in gpu
    hipMemcpy((void**)&d_num,sizeof(int))
    hipMemcpy((void**)&d_p, size * sizeof(int))

    //copy host to device[gpu]
    hipMemcpy(d_num, num, sizeof(int), hipMemcpyHostToDevice)
    hipMemcpy(p_num, p, size * sizeof(int), hipMemcpyHostToDevice)

    prime<<<1,size>>>(d_num,d_p,size);

    //copy device to host
    hipMemcpy(p, num, size * sizeof(int), hipMemcpyDeviceToHost)

    for (int i = 2; i <size; i++)
        cout<<i<<endl;
    
    hipFree(d_num);
    hipFree(d_p);
    return 0;
}