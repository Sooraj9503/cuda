#include "hip/hip_runtime.h"

#include <stdio.h>

//cuda kernel to add two integers
__global__ void addIntegers(int *a, int *b, int *device_result)
{
        *device_result = *a + *b;
}

int main()
{
        //host variables
        int host_a = 5;
        int host_b = 5;
        int host_result = 0;

        //device variables
        int *device_a, *device_b, *device_result;

        //allocate memory on the device
        hipMalloc((void**)&device_a, sizeof(int));
        hipMalloc((void**)&device_b, sizeof(int));
        hipMalloc((void**)&device_result, sizeof(int));

        //copy data from host to device
        hipMemcpy(device_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_b, &host_b, sizeof(int), hipMemcpyHostToDevice);

        //launch the kernel with
        addIntegers<<<1,1>>>(device_a, device_b, device_result);

        //copy result from device to host
        hipMemcpy(&host_result, device_result, sizeof(int), hipMemcpyDeviceToHost);

        printf("Result: %d\n", host_result);

        //free memory on the device
        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_result);

        return 0;

}

