
#include <hip/hip_runtime.h>
#include <stdio.h>

#define ROWS 4
#define COLS 6



int main() {
    int *arr1, *arr2, *result;
    int i, j;

    // Allocate memory on host
    arr1 = (int *)malloc(ROWS * COLS * sizeof(int));
    arr2 = (int *)malloc(ROWS * COLS * sizeof(int));
    result = (int *)malloc(ROWS * COLS * sizeof(int));

    // Initialize arrays
    for (i = 0; i < ROWS; i++) 
    {
        for (j = 0; j < COLS; j++) 
        {
            arr1[i * COLS + j] = i * COLS + j + 1;
            arr2[i * COLS + j] = i * COLS + j + 1 + ROWS * COLS;
        }
    }
    //Addition of both arrays
    for (i = 0; i < ROWS; i++) 
    {
        for (j = 0; j < COLS; j++) 
        {
            result[i * COLS + j] = arr1[i * COLS + j] + arr2[i * COLS + j];
        }
    }

    //Display 1st array
    printf("\nArray 1:\n");
    for (i = 0; i < ROWS; i++)
    {
        for (j = 0; j < COLS; j++)
        {
            printf("%3d ", arr1[i * COLS + j]);
        }
        printf("\n");
    }

    //Display 2nd array
    printf("\nArray 2:\n");
    for (i = 0; i < ROWS; i++)
    {
        for (j = 0; j < COLS; j++)
        {
            printf("%3d ", arr2[i * COLS + j]);
        }
        printf("\n");
    }

    // Print result
    printf("\nResult:\n");
    for (i = 0; i < ROWS; i++) {
        for (j = 0; j < COLS; j++) {
            printf("%3d ", result[i * COLS + j]);
        }
        printf("\n");
    }

    // Free memory
    free(arr1);
    free(arr2);
    free(result);

    return 0;
}