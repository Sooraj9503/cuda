#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void grandChild()
{
    printf(" CDAC\n");
}

__global__ void child()
{
    printf(" from");
    grandChild<<<1,1>>>();
}

__global__ void parent()
{
    printf("Hello");
    child<<<1,1>>>();
}

int main()
{
    parent<<<1,1>>>();
    hipDeviceSynchronize();

    return 0;
}