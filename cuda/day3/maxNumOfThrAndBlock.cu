#include<iostream>
#include<hip/hip_runtime.h>

int main()
{
    int maxThreadsPerBlock, maxBlocks;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocks, hipDeviceAttributeMaxGridDimX, 0);

    std::cout << "Maximum Threads per Block: "<< maxThreadsPerBlock << std::endl;
    std::cout << "Maximum Blocks: " << maxBlocks << std::endl;

    return 0;
}