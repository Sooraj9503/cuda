#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void threadInfo()
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	printf("ThreadIdx: %d, blockIdx: %d, BlockDim: %d, Effective Thread ID: %d\n", threadIdx.x, blockIdx.x, blockDim.x, threadID);

}

int main()
{
	int numBlocks = 3;
	int threadsPerBlock = 4;
	
	threadInfo<<<numBlocks, threadsPerBlock>>>();
	hipDeviceSynchronize();
	
	return 0;

}
