
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<chrono>
#define n 5

int** allocate_matrix(int rows, int cols) {
    int** matrix = (int**)malloc(rows * sizeof(int*));
    for(int i = 0; i < rows; i++) {
        matrix[i] = (int*)malloc(cols * sizeof(int));
    }
    return matrix;
}

void initialize_matrix(int** matrix, int rows, int cols) {
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            matrix[i][j] = rand() % 10;
        }
    }
}

void print_matrix(int** matrix, int rows, int cols) {
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            printf("%d ", matrix[i][j]);
        }
        printf("\n");
    }
}

void matrix_multiply(int** A, int** B, int** C) {
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            C[i][j] = 0;
            for(int k = 0; k < n; k++) {
                C[i][j] += A[i][k] * B[k][j];
            }
        }
    }
}

int main() {
    int** A = allocate_matrix(n, n);
    int** B = allocate_matrix(n, n);
    int** C = allocate_matrix(n, n);

    initialize_matrix(A, n, n);
    initialize_matrix(B, n, n);

    auto cpu_start_time = std::chrono::high_resolution_clock::now();
    matrix_multiply(A, B, C);
    auto cpu_end_time = std::chrono::high_resolution_clock::now();

    auto cpu_duration_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(cpu_end_time - cpu_start_time).count();
    
    // print matrices A, B, and C
    printf("Matrix A:\n");
    print_matrix(A, n, n);
    printf("Matrix B:\n");
    print_matrix(B, n, n);
    printf("\nMatrix C(Result of A*B):\n");
    print_matrix(C, n, n);
    printf("Time taken by CPU: %f microseconds\n", (float)cpu_duration_ns / 1000);

    for(int i = 0; i < n; i++)
    {
        free(A[i]);
        free(B[i]);
        free(C[i]);
    }
    free(A);
    free(B);
    free(C);
    return 0;
}