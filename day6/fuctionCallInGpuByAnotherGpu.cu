#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void child()
{
    printf(" World\n");
}

__global__ void parent()
{
    printf("Hello ");
    child<<<1,1>>>();
}

int main()
{    
    parent<<<1,1>>>();
    hipDeviceSynchronize();

    return 0;
}