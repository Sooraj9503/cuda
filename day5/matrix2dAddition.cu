#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 5

__global__ void add(int *a, int *b, int *c) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N)
    {
    c[i*N+j] = a[i*N+j] + b[i*N+j];
    }
}

int main() 
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    // Allocate space for arrays a, b, and c
    a = (int *)malloc(N * N * sizeof(int));
    b = (int *)malloc(N * N * sizeof(int));
    c = (int *)malloc(N * N * sizeof(int));

    // Allocate memory on the GPU
    hipMalloc((void **)&d_a, N * N * sizeof(int));
    hipMalloc((void **)&d_b, N * N * sizeof(int));
    hipMalloc((void **)&d_c, N * N * sizeof(int));

    // Initialize arrays a and b
    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++) 
        {
                a[i*N+j] = 1;
                b[i*N+j] = 1;
        }
    }
     for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) 
        {
                printf("%d ", c[i*N+j]);
        }
        printf("\n");
    }

    // Copy arrays a and b to the GPU
    hipMemcpy(d_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU
    int threadsPerBlock = 5;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

    // Copy the result back to the CPU
    hipMemcpy(c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) 
        {
                printf("%d ", c[i*N+j]);
        }
        printf("\n");
    }

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}