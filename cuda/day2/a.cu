#include<hip/hip_runtime.h>

#include<stdio.h>

int main()
{
    const int arraySize = 5;

    //host (cpu) data
    float hostArray[arraySize] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float hostResult[arraySize];

    //Device (gpu) data
    float* deviceArray;
    hipMalloc((void**)&deviceArray, arraySize * sizeof(float));

    //copy data from cpu to gpu
    hipMemcpy(deviceArray, hostArray, arraySize * sizeof(float), hipMemcpyHostToDevice);

    //copy data from gpu to cpu
    hipMemcpy(hostResult, deviceArray, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    //Display results using printf
    printf("Original array: ");
    for(int i = 0; i < arraySize; ++i)
    {
        printf("%f ", hostArray[i]);
    }
    printf("\n");
    printf("Copied array: ");
    for(int i = 0; i < arraySize; ++i)
    {
        printf("%f ", hostResult[i]);
    }
    printf("\n");

    //free allocated memory on gpu
    hipFree(deviceArray);
    return 0;
}
