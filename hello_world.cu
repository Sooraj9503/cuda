#include "hip/hip_runtime.h"
{
  "nbformat": 4,
  "nbformat_minor": 0,
  "metadata": {
    "colab": {
      "provenance": [],
      "gpuType": "T4",
      "authorship_tag": "ABX9TyNi9xN+A/V/wB29S4MqUNqO",
      "include_colab_link": true
    },
    "kernelspec": {
      "name": "python3",
      "display_name": "Python 3"
    },
    "language_info": {
      "name": "python"
    },
    "accelerator": "GPU"
  },
  "cells": [
    {
      "cell_type": "markdown",
      "metadata": {
        "id": "view-in-github",
        "colab_type": "text"
      },
      "source": [
        "<a href=\"https://colab.research.google.com/github/Sooraj9503/cuda/blob/main/hello_world.cu\" target=\"_parent\"><img src=\"https://colab.research.google.com/assets/colab-badge.svg\" alt=\"Open In Colab\"/></a>"
      ]
    },
    {
      "cell_type": "code",
      "execution_count": 2,
      "metadata": {
        "colab": {
          "base_uri": "https://localhost:8080/"
        },
        "id": "TdxMf8Jy-XLl",
        "outputId": "9ba54575-1f89-4494-ca37-79c94ac13358"
      },
      "outputs": [
        {
          "output_type": "stream",
          "name": "stdout",
          "text": [
            "Writing hello_world.cu\n"
          ]
        }
      ],
      "source": [
        "%%writefile hello_world.cu\n",
        "\n",
        "#include <iostream>\n",
        "\n",
        "__global__ void helloFromGPU() {\n",
        "    printf(\"Hello World from GPU!\\n\");\n",
        "}\n",
        "\n",
        "int main() {\n",
        "    helloFromGPU<<<1, 1>>>();\n",
        "    hipDeviceSynchronize();\n",
        "    std::cout << \"Hello World from CPU!\\n\";\n",
        "    return 0;\n",
        "}\n"
      ]
    },
    {
      "cell_type": "code",
      "source": [
        "!nvcc hello_world.cu -o hello_world\n"
      ],
      "metadata": {
        "id": "oeG6PSLv_Ffn"
      },
      "execution_count": 3,
      "outputs": []
    },
    {
      "cell_type": "code",
      "source": [
        "!./hello_world\n"
      ],
      "metadata": {
        "colab": {
          "base_uri": "https://localhost:8080/"
        },
        "id": "a4xRfIZj_KZE",
        "outputId": "87b85d46-fe25-486a-948c-04d7f02936ef"
      },
      "execution_count": 4,
      "outputs": [
        {
          "output_type": "stream",
          "name": "stdout",
          "text": [
            "Hello World from GPU!\n",
            "Hello World from CPU!\n"
          ]
        }
      ]
    }
  ]
}