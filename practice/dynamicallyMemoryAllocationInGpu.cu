
#include <hip/hip_runtime.h>
#include <stdio.h>
#define ROWS 4
#define COLS 6

 //Kernel function for addition
    __global__ void add(int *dev_arr1, int *dev_arr2, int *dev_result)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx < ROWS * COLS)
        {
            dev_result[idx] = dev_arr1[idx] + dev_arr2[idx];
        }
    }

int main() {
    int *arr1, *arr2, *result;
    int *dev_arr1, *dev_arr2, *dev_result;
    int i, j;

    // Allocate memory on host
    arr1 = (int *)malloc(ROWS * COLS * sizeof(int));
    arr2 = (int *)malloc(ROWS * COLS * sizeof(int));
    result = (int *)malloc(ROWS * COLS * sizeof(int));

    // Initialize arrays
    for (i = 0; i < ROWS; i++) 
    {
        for (j = 0; j < COLS; j++) 
        {
            arr1[i * COLS + j] = i * COLS + j + 1;
            arr2[i * COLS + j] = i * COLS + j + 1 + ROWS * COLS;
        }
    }

    //Allocate memory on gpu
    hipMalloc((void **)&dev_arr1, ROWS * COLS * sizeof(int));
    hipMalloc((void **)&dev_arr2, ROWS * COLS * sizeof(int));
    hipMalloc((void **)&dev_result, ROWS * COLS * sizeof(int));

    //Copy data from host to device
    hipMemcpy( dev_arr1, arr1, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( dev_arr2, arr2, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);

    //Kernel function call
    int threads = 256;
    int blocks = (ROWS * COLS + threads - 1) / threads;

    add<<<blocks, threads>>>(dev_arr1, dev_arr2, dev_result);

    //Copy data from device to host
    hipMemcpy(result, dev_result, ROWS * COLS * sizeof(int), hipMemcpyDeviceToHost);

    //Display 1st array
    printf("\nArray 1:\n");
    for (i = 0; i < ROWS; i++)
    {
        for (j = 0; j < COLS; j++)
        {
            printf("%3d ", arr1[i * COLS + j]);
        }
        printf("\n");
    }

    //Display 2nd array
    printf("\nArray 2:\n");
    for (i = 0; i < ROWS; i++)
    {
        for (j = 0; j < COLS; j++)
        {
            printf("%3d ", arr2[i * COLS + j]);
        }
        printf("\n");
    }

    // Print result
    printf("\nResult:\n");
    for (i = 0; i < ROWS; i++) {
        for (j = 0; j < COLS; j++) {
            printf("%3d ", result[i * COLS + j]);
        }
        printf("\n");
    }

    // Free memory
    free(arr1);
    free(arr2);
    free(result);
    hipFree(dev_arr1);
    hipFree(dev_arr2);
    hipFree(dev_result);
    
    return 0;
}